
#include <hip/hip_runtime.h>
#include <stdio.h>

// Define the CUDA kernel
__global__ void vector_add(float *a, float *b, float *c, int N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    int N = 5;
    float *A, *B, *C; // host   
    float *d_A, *d_B, *d_C;  // device

    // Allocate host memory
    A = (float*)malloc(N * sizeof(float));
    B = (float*)malloc(N * sizeof(float));
    C = (float*)malloc(N * sizeof(float));

    // Allocate device memory
    hipMalloc((void**)&d_A, N * sizeof(float));
    hipMalloc((void**)&d_B, N * sizeof(float));
    hipMalloc((void**)&d_C, N * sizeof(float));

    // Initialize host arrays and copy to device
    for (int i = 0; i < N; ++i) {
        A[i] = float(i);
        B[i] = float(i);
    }
    hipMemcpy(d_A, A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * sizeof(float), hipMemcpyHostToDevice);

    // Launch the CUDA kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vector_add<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N); // <<<...>>> syntax is used to specify the number of blocks and threads per block.

    // Copy result back to host
    hipMemcpy(C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);

    // Print the result
    for (int i = 0; i < N; i++) {
        printf("%f + %f : %f", A[i], B[i], C[i]);
        printf("\n");
    }

    // Cleanup
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    free(A); free(B); free(C);

    return 0;
}
