
#include <hip/hip_runtime.h>
#include <stdio.h>

// Define the CUDA kernel
__global__ void matAdd(float *a, float *b, float *c, int P, int Q) {
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (row < P && col < Q) {
        c[row * Q + col] = a[row * Q + col] + b[row * Q + col];
    }
}

int main() {
    int P=200000, Q=300000; // high values give segmentation fault
    int N = Q;
    float *h_A, *h_B, *h_C;     // host
    float *d_A, *d_B, *d_C;     // device
    // Allocate host memory
    h_A = (float *)malloc(P * Q * sizeof(float));
    h_B = (float *)malloc(P * Q * sizeof(float));
    h_C = (float *)malloc(P * Q * sizeof(float));

    // Allocate device memory
    hipMalloc((void **)&d_A, P * Q * sizeof(float));
    hipMalloc((void **)&d_B, P * Q * sizeof(float));
    hipMalloc((void **)&d_C, P * Q * sizeof(float));
    
    // initialize host matrices & copy to device
    for (int i=0; i < P; i++) {
        for (int j=0; j<Q; j++) {
            h_A[i * Q + j] = float(i);
            h_B[i * Q + j] = float(j);
            h_C[i * Q + j] = 0.0f; // initialize to 0
        }
    }
    hipMemcpy(d_A, h_A, P * Q * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, P * Q * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, P * Q * sizeof(float), hipMemcpyHostToDevice);

    // for every row, invoke kernel
    
    // int threadsPerBlock = 256;
    // int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    
    dim3 blockDim(16, 16); // threadsPerBlock: 256
    dim3 gridDim((P + blockDim.x - 1)/blockDim.x, (Q + blockDim.y - 1)/blockDim.y);

    // calculating kernel runtime
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    
    // for (int i=0; i < P; i++) {
    //         float *d_A_row = &d_A[i * Q];
    //         float *d_B_row = &d_B[i * Q];
    //         float *d_C_row = &d_C[i * Q];
    //         // invoking kernel
    //         matAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A_row, d_B_row, d_C_row, Q);
    // } 
    
    // invoking kernel
    matAdd<<<gridDim, blockDim>>>(d_A, d_B, d_C, P, Q);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipMemcpy(h_C, d_C, P * Q * sizeof(float), hipMemcpyDeviceToHost);

    // print
    // for (int i=0; i < P; i++) {
    //     for (int j=0; j<Q; j++) {
    //         printf("%f", h_A[i * Q + j]);
    //     }
    //     printf("\n");
    // }
    // printf("\n\n");
    // for (int i=0; i < P; i++) {
    //     for (int j=0; j<Q; j++) {
    //         printf("%f", h_B[i * Q + j]);
    //     }
    //     printf("\n");
    // }
    // printf("\n\n");
    // for (int i=0; i < P; i++) {
    //     for (int j=0; j<Q; j++) {
    //         printf("%f", h_C[i * Q + j]);
    //     }
    //     printf("\n");
    // }

    // time taken
    float elapsed_time = 0.0f;
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("\nTime taken: %f ms\n", elapsed_time);
    

    // Cleanup
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    free(h_A); free(h_B); free(h_C);
    hipEventDestroy(start); hipEventDestroy(stop);

    return 0;
}