
#include <hip/hip_runtime.h>
#include <stdio.h>

// Define the CUDA kernel
__global__ void matAdd(float *a, float *b, float *c, int N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    int P=2, Q=3;
    int N = Q;
    float *h_A, *h_B, *h_C;     // host
    float *d_A, *d_B, *d_C;     // device
    // Allocate host memory
    h_A = (float *)malloc(P * Q * sizeof(float));
    h_B = (float *)malloc(P * Q * sizeof(float));
    h_C = (float *)malloc(P * Q * sizeof(float));

    // Allocate device memory
    hipMalloc((void **)&d_A, P * Q * sizeof(float));
    hipMalloc((void **)&d_B, P * Q * sizeof(float));
    hipMalloc((void **)&d_C, P * Q * sizeof(float));

    // initialize host matrices & copy to device
    for (int i=0; i < P; i++) {
        for (int j=0; j<Q; j++) {
            h_A[i * Q + j] = float(i);
            h_B[i * Q + j] = float(j);
            h_C[i * Q + j] = 0.0f; // initialize to 0
        }
    }
    hipMemcpy(d_A, h_A, P * Q * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, P * Q * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, P * Q * sizeof(float), hipMemcpyHostToDevice);

    // for every row, invoke kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    for (int i=0; i < P; i++) {
            float *d_A_row = &d_A[i * Q];
            float *d_B_row = &d_B[i * Q];
            float *d_C_row = &d_C[i * Q];
            // invoking kernel
            matAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A_row, d_B_row, d_C_row, Q);
    }
    hipMemcpy(h_C, d_C, P * Q * sizeof(float), hipMemcpyDeviceToHost);

    // print
    for (int i=0; i < P; i++) {
        for (int j=0; j<Q; j++) {
            printf("%f", h_A[i * Q + j]);
        }
        printf("\n");
    }
    printf("\n\n");
    for (int i=0; i < P; i++) {
        for (int j=0; j<Q; j++) {
            printf("%f", h_B[i * Q + j]);
        }
        printf("\n");
    }
    printf("\n\n");
    for (int i=0; i < P; i++) {
        for (int j=0; j<Q; j++) {
            printf("%f", h_C[i * Q + j]);
        }
        printf("\n");
    }
    

    // Cleanup
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    free(h_A); free(h_B); free(h_C);

    return 0;
}