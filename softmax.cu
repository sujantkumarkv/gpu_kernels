
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void exponentiate (float* a, int N) {
    // softmax kernel
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    // exponentiated
    if (i < N) {
        a[i] = __expf(a[i]);
    }
}

__global__ void reduction (float* a, float* sum, int N) {
    // calculate thread ID
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // boundary check
    if (i < N) {
        atomicAdd(sum, a[i]);
    }
}

__global__ void softmax (float* a, float* sum, int N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    // boundary check
    if (i < N) {
        a[i] /= *sum;
    }
}

int main() {
    int N = 6;
    float *h_A; // cpu
    float *d_A, *d_sum; // gpu

    // memory allocation
    // host
    h_A = (float *)malloc(N * sizeof(float));
    //device
    hipMalloc((void **)&d_A, N * sizeof(float));
    hipMalloc((void **)&d_sum, sizeof(float));

    float h_sum = 0.0f;
    // initialize host vectors & copy to device
    for (int i=1; i < N; i++) {
        h_A[i] = float(i);
    }
    // print initially
    printf("A:\n");
    for (int i=1; i < N; i++) {
        printf("%f ", h_A[i]);
    }
    printf("\n");

    hipMemcpy(d_A, h_A, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_sum, &h_sum, sizeof(float), hipMemcpyHostToDevice);

    // calculating kernel runtime
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    // launch kernels
    exponentiate<<< 1, N >>>(d_A, N);
    reduction<<< 1, N >>>(d_A, d_sum, N);
    softmax<<< 1, N >>>(d_A, d_sum, N);

    // stop timer
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    // copy result back
    hipMemcpy(h_A, d_A, N * sizeof(float), hipMemcpyDeviceToHost);
    // print
    printf("Softmax(A):\n");
    for (int i=1; i < N; i++) {
        printf("%f ", h_A[i]);
    }
    printf("\n");
    // time taken
    float elapsed_time = 0.0f;
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("\nTime taken: %f ms\n", elapsed_time);
    printf("\n");
    // Cleanup
    hipFree(d_A); hipFree(d_sum);
    free(h_A);
    return 0;
}