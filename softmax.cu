#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void exponentiate (float* a, int N) {
    // softmax kernel
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    // exponentiated
    if (i < N) {
        a[i] = __expf(a[i]);
    }
}

__global__ void reduction (float* a, float* sum, int N) {
    // calculate thread ID
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // boundary check
    if (i < N) {
        atomicAdd(sum, a[i]);
    }
}

__global__ void softmax (float* a, float* sum) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    // boundary check
    if (i < N) {
        a[i] /= sum;
    }
}

int main() {
    int N = 6;
    float *h_A; // cpu
    float *d_A, *d_sum; // gpu

    // memory allocation
    // host
    h_A = (float *)malloc(N * sizeof(float));
    //device
    d_A = hipMalloc((void **)&d_A, N * sizeof(float));
    d_sum = hipMalloc((void **)&d_sum, sizeof(float));

    float h_sum = 0.0f;
    // initialize host vectors & copy to device
    for (int i=1; i < N; i++) {
        h_A[i] = float(i);
    }
    // print initially
    printf("A:\n");
    for (int i=1; i < N; i++) {
        printf("%f ", h_A[i]);
    }
    printf("\n");

    hipMemcpy(d_A, h_A, N*sizeof(float), hipMemcpyHostToDevice);
    float h_sum = 0.0f;
    hipMemcpy(d_sum, &h_sum, sizeof(float), hipMemcpyHostToDevice);


    // launch kernels
    exponentiate<<< 1, N >>>(d_A, N);
    reduction<<< 1, N >>>(d_A, d_sum, N);
    softmax<<< 1, N >>>(d_A, d_sum);

    // copy result back
    hipMemcpy(h_A, d_A, 1 * sizeof(float), hipMemcpyDeviceToHost);
    // print
    for (int i=1; i < N; i++) {
        printf("softmax(A):\n");
        printf("% ", h_A[i]);
    }
    // Cleanup
    hipFree(d_A); hipFree(d_sum);
    free(h_A);
    return 0;
}