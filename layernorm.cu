
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void mean (float* a, float* sum, float* mean, int N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N) {
        atomicAdd(sum, a[i]);
    }
    __syncthreads();
    *mean = *sum / N;
}

__global__ void variance (float* a, float* mean, float* var, int N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    __shared__ float sum;
    if (i == 0) {
        sum = 0.0f;
    }
    __syncthreads();
    if (i < N) {
        atomicAdd(&sum, powf((a[i] - *mean), 2));
    }
    __syncthreads();
    if (i == 0) {
        *var = sum / (N-1);
    }
}

__global__ void layernorm (float* a, float* mean, float* var, float* layernorm, int N) {
    // constants, learnable parameters
    float epsilon = 1e-8, gamma = 1.0f, beta = 0.0f;
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N) {
        layernorm[i] = ((a[i] - *mean) / sqrtf(*var + epsilon)) * gamma + beta;
    }
}

int main() {
    int N = 6;
    float *h_A, *h_mean, *h_var, *h_layernorm; // cpu
    float *d_A, *d_sum, *d_mean, *d_var, *d_layernorm; // gpu

    // memory allocation
    // host
    h_A = (float *)malloc(N * sizeof(float));
    h_mean = (float *)malloc(1 * sizeof(float));
    h_var = (float *)malloc(1 * sizeof(float)); 
    h_layernorm = (float *)malloc(N * sizeof(float));
    //device
    hipMalloc((void **)&d_A, N * sizeof(float));
    hipMalloc((void **)&d_sum, 1 * sizeof(float));
    hipMalloc((void **)&d_mean, 1 * sizeof(float));
    hipMalloc((void **)&d_var, 1 * sizeof(float));
    hipMalloc((void **)&d_layernorm, N * sizeof(float));

    // initialize host vectors & copy to device
    for (int i=1; i < N; i++) {
        h_A[i] = float(i);
    }
    // print initially
    printf("A:\n");
    for (int i=1; i < N; i++) {
        printf("%f ", h_A[i]);
    }
    printf("\n");

    // later, atomicAdd
    float h_sum = 0.0f;
    hipMemcpy(d_sum, &h_sum, 1*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_A, h_A, N*sizeof(float), hipMemcpyHostToDevice);
    

    // calculating kernel runtime
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // start
    hipEventRecord(start, 0);

    // launch kernels
    mean<<< 1, N >>>(d_A, d_sum, d_mean, N);
    variance<<< 1, N >>>(d_A, d_mean, d_var, N);
    layernorm<<< 1, N >>>(d_A, d_mean, d_var, d_layernorm, N);

    // stop timer
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    // copy result back
    hipMemcpy(h_A, d_A, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_mean, d_mean, 1 * sizeof(float), hipMemcpyDeviceToHost); 
    hipMemcpy(h_var, d_var, 1 * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_layernorm, d_layernorm, N * sizeof(float), hipMemcpyDeviceToHost);
    
    // print
    /*
    h_mean & h_var below gives error & thus *h_mean and *h_var is used bcz
    printf expects a double/float for %f format, but a float* (pointer to float) was given.
    and, h_A[i] and h_layernorm[i] are not pointers, they are float values, so they work :)
    */
    printf("Mean: %f\n", *h_mean);
    printf("Variace: %f\n", *h_var);
    printf("Layernorm:\n");
    for (int i=1; i < N; i++) {
        printf("%f ", h_layernorm[i]);
    }
    printf("\n");
    // time taken
    float elapsed_time = 0.0f;
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("\nFor N: %d, Time taken: %f ms\n", N, elapsed_time);
    printf("\n");
    // Cleanup
    hipFree(d_A); hipFree(d_sum);
    free(h_A);
    return 0;
}